#include "utils.h"
#include <hipblas.h>
#include <iostream>
#include <vector>
#define ALIGN 32

int cudaErrorHandle(hipError_t result)
{
    if (result = hipSuccess)
    {
        return 0;
    }
    else if (result == hipErrorInvalidValue)
    {
        std::cout << "Error: Invalid value\n";
        return 1;
    }
    else if (result == hipErrorInvalidMemcpyDirection)
    {
        std::cout << "Error: Invalid memory copy direction\n";
        return 1;
    }
}

int memcpyPinned(hipComplex *h_A, hipComplex *h_B, hipComplex *h_C, hipComplex *h_alpha, hipComplex *h_beta, const cf_t *A,
                 const cf_t *B, const cf_t *C, const cf_t *alpha, const cf_t *beta, const int M, const int N,
                 const int K)
{
    if (sizeof(cf_t) != sizeof(hipComplex))
    {
        return 1;
    }
    memcpy(h_A, A, sizeof(cf_t) * M * K);
    memcpy(h_B, B, sizeof(cf_t) * K * N);
    memcpy(h_C, C, sizeof(cf_t) * M * N);
    h_alpha->x = alpha->r;
    h_alpha->y = alpha->i;
    h_beta->x = beta->r;
    h_beta->y = beta->i;
}

std::pair<int, float> Arrays2Device(hipComplex *d_A, hipComplex *d_B, hipComplex *d_C, hipComplex *h_A, hipComplex *h_B,
                                    hipComplex *h_C, int M, int N, int K)
{
    int check = 0;
    hipError_t result;

    // 初期化
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    result = hipMemcpy(d_A, h_A, sizeof(hipComplex) * M * K, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);
    hipEventRecord(start);
    result = hipMemcpy(d_B, h_B, sizeof(hipComplex) * K * N, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    check += cudaErrorHandle(result);
    result = hipMemcpy(d_C, h_C, sizeof(hipComplex) * M * N, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);
    return std::make_pair(check, milliseconds);
}

std::pair<int, float> Array2Host(hipComplex *h_A, hipComplex *h_B, hipComplex *h_C, hipComplex *d_A, hipComplex *d_B,
                                 hipComplex *d_C, int M, int N, int K)
{
    int check = 0;
    hipError_t result;

    // 初期化
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    result = hipMemcpy(h_A, d_A, sizeof(hipComplex) * M * K, hipMemcpyDeviceToHost);
    check += cudaErrorHandle(result);
    result = hipMemcpy(h_B, d_B, sizeof(hipComplex) * K * N, hipMemcpyDeviceToHost);
    check += cudaErrorHandle(result);
    hipEventRecord(start);
    result = hipMemcpy(h_C, d_C, sizeof(hipComplex) * M * N, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    check += cudaErrorHandle(result);
    return std::make_pair(check, milliseconds);
}

int main(int argc, char *argv[])
{
    if (argc != 4)
    {
        std::cerr << "Usage: " << argv[0] << " <rows> <cols>" << std::endl;
        return 1;
    }
    int N = atoi(argv[0]);
    int M = atoi(argv[1]);
    int K = atoi(argv[2]);
    int iters = atoi(argv[3]);

    // initialize host arrays
    cf_t *A = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * K);
    cf_t *B = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * K * N);
    cf_t *C = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * N);
    hipComplex *h_A, *h_B, *h_C;
    hipHostAlloc((void **)h_A, sizeof(hipComplex) * M * K, hipHostMallocDefault);
    hipHostAlloc((void **)h_B, sizeof(hipComplex) * K * N, hipHostMallocDefault);
    hipHostAlloc((void **)h_C, sizeof(hipComplex) * M * N, hipHostMallocDefault);
    cf_t alpha, beta;
    hipComplex d_alpha, d_beta;

    // initialize device arrays
    hipComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)d_A, sizeof(hipComplex) * M * K);
    hipMalloc((void **)d_B, sizeof(hipComplex) * K * N);
    hipMalloc((void **)d_C, sizeof(hipComplex) * M * N);

    // initialize results
    std::vector<double> ms_results, memcpy_d2h_results, memcpy_h2d_results;
    hipEvent_t start, stop;

    // 初期化
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize cuda, cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // warm-up run
    float warmup;
    setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
    memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
    Arrays2Device(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K);
    hipEventRecord(start);
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&warmup, start, stop);
    Array2Host(h_A, h_B, h_C, d_A, d_B, d_C, M, N, K);

    for (int i = 0; i < iters; i++)
    {
        setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
        memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
        auto mem_h2d_result = Arrays2Device(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K);
        memcpy_h2d_results.push_back(mem_h2d_result.second);
        hipEventRecord(start);
        hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        auto mem_d2h_result = Array2Host(h_A, h_B, h_C, d_A, d_B, d_C, M, N, K);
        memcpy_d2h_results.push_back(mem_d2h_result.second);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        ms_results.push_back(milliseconds);
    }
    std::cout << getMean(ms_results) << "," << getStdev(ms_results);
    std::cout << getMean(memcpy_h2d_results) << "," << getStdev(memcpy_h2d_results);
    std::cout << getMean(memcpy_d2h_results) << "," << getStdev(memcpy_d2h_results);
    std::cout << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    free(A);
    free(B);
    free(C);
    hipblasDestroy(handle);
    return 0;
}