#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// ホストから呼び出される関数
extern "C" void addVectors(int* c, const int* a, const int* b, int size) {
    int* d_a;
    int* d_b;
    int* d_c;

    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<(size + 255) / 256, 256>>>(d_c, d_a, d_b, size);

    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

/*
cuda shared library compile
nvcc -shared -o libcuda_add.so kernel.cu
*/
