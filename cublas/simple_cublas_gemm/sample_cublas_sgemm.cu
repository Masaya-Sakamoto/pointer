
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdlib>
#include <hipblas.h>
#include <iostream>

#define DEBUG
#define INFO
#define CBLAS

#ifdef CBLAS
#include <cblas.h>
#endif

#define M 2
#define N 61440
#define K 3200
#define ALIGN 64

void initArray(size_t elements, float *array)
{
    srand(0); // Seed for reproducibility
    for (size_t i = 0; i < elements; ++i)
    {
        array[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

void print_matrix(const float *A, int rows, int cols)
{
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            std::cout << A[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main()
{
    float *A, *B, *C;
    float *t_A, *t_B, *t_C; // for cblas(cblas_col_major)
    float *h_A, *h_B, *h_C;

    // Allocate main memory (aligned)
    A = (float *)aligned_alloc(ALIGN, M * K * sizeof(float));
    B = (float *)aligned_alloc(ALIGN, K * N * sizeof(float));
    C = (float *)aligned_alloc(ALIGN, M * N * sizeof(float));
    t_A = (float *)aligned_alloc(ALIGN, M * K * sizeof(float));
    t_B = (float *)aligned_alloc(ALIGN, K * N * sizeof(float));
    t_C = (float *)aligned_alloc(ALIGN, M * N * sizeof(float));

    // Allocate page-locked host memory
    hipHostAlloc((void **)&h_A, M * K * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, K * N * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, M * N * sizeof(float), hipHostMallocDefault);

    // Initialize arrays
    initArray(M * K, A);
    initArray(K * N, B);
    initArray(M * N, C);

    // Copy to page-locked host buffers
    memcpy(t_A, A, M * K * sizeof(float));
    memcpy(t_B, B, K * N * sizeof(float));
    memcpy(t_C, C, M * N * sizeof(float));
    memcpy(h_A, A, M * K * sizeof(float));
    memcpy(h_B, B, K * N * sizeof(float));
    memcpy(h_C, C, M * N * sizeof(float));

    // GPU initialization
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipblasSetStream(handle, stream); // Associate cuBLAS with stream

    float *d_A, *d_B, *d_C;
    const float d_alpha = 1.0f;
    const float d_beta = 0.0f;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // Warm-up: Async data transfer and SGEMM
    hipMemcpyAsync(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
    hipStreamSynchronize(stream);

    // Reinitialize data
    initArray(M * K, A);
    initArray(K * N, B);
    initArray(M * N, C);
    memcpy(t_A, A, M * K * sizeof(float));
    memcpy(t_B, B, K * N * sizeof(float));
    memcpy(t_C, C, M * N * sizeof(float));
    memcpy(h_A, A, M * K * sizeof(float));
    memcpy(h_B, B, K * N * sizeof(float));
    memcpy(h_C, C, M * N * sizeof(float));

#ifdef DEBUG
    // CPU timing
    std::cout << "CPU: calculation";
    fflush(stdout);
    auto cpu_start = std::chrono::high_resolution_clock::now();
#ifdef DEBUG
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M, N, K, 1.0f, A, K, B, N, 0.0f, C, N);
#endif
    auto cpu_duration = std::chrono::high_resolution_clock::now() - cpu_start;
    auto cpu_us = std::chrono::duration_cast<std::chrono::microseconds>(cpu_duration).count();
    std::cout << " done!  " << cpu_us << " μs\n";
#endif

#ifdef DEBUG
    // CPU timing
    std::cout << "CPU: calculation";
    fflush(stdout);
    cpu_start = std::chrono::high_resolution_clock::now();
#ifdef DEBUG
    cblas_sgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, N, M, K, 1.0f, t_B, N, t_A, K, 0.0f, t_C, N);
#endif
    cpu_duration = std::chrono::high_resolution_clock::now() - cpu_start;
    cpu_us = std::chrono::duration_cast<std::chrono::microseconds>(cpu_duration).count();
    std::cout << " done!  " << cpu_us << " μs\n";
#endif

#ifdef DEBUG
    // GPU timing (kernel + result copy-back)
    std::cout << "GPU: calculation";
    fflush(stdout);
    auto gpu_start = std::chrono::high_resolution_clock::now();
#endif

    // Pre-copy all data to GPU (excluded from timing)
    hipMemcpyAsync(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
    hipMemcpyAsync(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

#ifdef DEBUG
    auto gpu_duration = std::chrono::high_resolution_clock::now() - gpu_start;
    auto gpu_us = std::chrono::duration_cast<std::chrono::microseconds>(gpu_duration).count();
    std::cout << " done!  " << gpu_us << " μs\n";
#endif

#ifdef DEBUG
    // Compare element-wise with tolerance
    bool results_match_gpu = true;
    bool results_match_cpuCol = true;
    bool results_match_gpu_cpuCol = true;
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            float cblas_val = C[i * N + j];
            float cblas_col_val = t_C[i * N + j];
            float cublas_val = h_C[i * N + j];

            // Use a tolerance to compare floating-point values
            if (fabsf(cblas_val - cublas_val) > 1e-3f)
            {
                results_match_gpu = false;
#ifdef INFO
                printf("Mismatch at position (%d, %d): CBLAS=%.4f vs cuBLAS=%.4f\n", i + 1, j + 1, cblas_val,
                       cublas_val);
#endif
            }
            if (fabsf(cblas_val - cblas_col_val) > 1e-3f)
            {
                results_match_cpuCol = false;
#ifdef INFO
                printf("Mismatch at position (%d, %d): CBLAS=%.4f vs CBLAS_COL=%.4f\n", i + 1, j + 1, cblas_val,
                       cublas_val);
#endif
            }
            if (fabsf(cblas_col_val - cublas_val) > 1e-3f)
            {
                results_match_gpu_cpuCol = false;
#ifdef INFO
                printf("Mismatch at position (%d, %d): CBLAS_COL=%.4f vs cuBLAS=%.4f\n", i + 1, j + 1, cblas_col_val,
                       cublas_val);
#endif
            }
        }
    }

    if (results_match_gpu)
    {
        printf("Results match!\n");
        for (int _chk = 0; _chk < 10; _chk++)
        {
            int chk_id = rand() % (N * M);
            printf("%3d, id=%9d, C=%7.4f, t_C=%7.4f\n", _chk, chk_id, C[chk_id], h_C[chk_id]);
        }
    }
    else
    {
        printf("Results do not match within tolerance.\n");
    }

    if (results_match_cpuCol)
    {
        printf("Results cpuCol match!\n");
        for (int _chk = 0; _chk < 10; _chk++)
        {
            int chk_id = rand() % (N * M);
            printf("%3d, id=%9d, C=%7.4f, t_C=%7.4f\n", _chk, chk_id, C[chk_id], t_C[chk_id]);
        }
    }
    else
    {
        printf("Results cpuCol do not match within tolerance.\n");
    }

    if (results_match_gpu_cpuCol)
    {
        printf("Results gpu_cpuCol match!\n");
        for (int _chk = 0; _chk < 10; _chk++)
        {
            int chk_id = rand() % (N * M);
            printf("%3d, id=%9d, C=%7.4f, t_C=%7.4f\n", _chk, chk_id, t_C[chk_id], h_C[chk_id]);
        }
    }
    else
    {
        printf("Results gpu_cpuCol do not match within tolerance.\n");
    }
#endif

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    free(A);
    free(B);
    free(C);
    free(t_A);
    free(t_B);
    free(t_C);
    hipblasDestroy(handle);
    hipStreamDestroy(stream);

    return 0;
}