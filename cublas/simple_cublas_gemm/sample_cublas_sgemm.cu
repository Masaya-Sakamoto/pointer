
#include <hip/hip_runtime.h>
#include <cblas.h>
#include <cstdlib>
#include <hipblas.h>
#include <iostream>

#define M 3
#define N 4
#define K 2
#define ALIGN 64

template <typename T> void initArray(size_t elements, T *array)
{
    srand(0); // Seed for reproducibility
    for (size_t i = 0; i < elements; ++i)
    {
        array[i] = static_cast<T>(rand()) / RAND_MAX;
    }
}

int main()
{
    float *A, *B, *C, *h_C;
    A = (float *)aligned_alloc(ALIGN, M * K * sizeof(float));
    B = (float *)aligned_alloc(ALIGN, K * N * sizeof(float));
    C = (float *)aligned_alloc(ALIGN, M * N * sizeof(float));
    h_C = (float *)aligned_alloc(ALIGN, M * N * sizeof(float)); // for dump
    // Initialize array
    initArray(M * K, A);
    initArray(K * N, B);
    initArray(M * N, C);
    initArray(M * N, h_C);

    // cblas
    cblas_sgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, M, N, K, 1.0f, A, M, B, K, 0.0f, C, M);

    // Initialize cuBLAS context.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // memcpy to device
    float *d_A, *d_B, *d_C;
    const float d_alpha = 1.0f;
    const float d_beta = 0.0f;
    hipMalloc((void **)&d_A, M * K * sizeof(float));
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_B, K * N * sizeof(float));
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_C, M * N * sizeof(float));
    hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice);
    // cuBLAS sgemm
    hipblasSgemm(
        handle, 
        HIPBLAS_OP_T, HIPBLAS_OP_T, // A, B 両方を転置
        N,                                // 結果行列の行数（実際はB^Tの行数）
        M,                                // 結果行列の列数（実際はA^Tの列数）
        K,                                // 内部次元
        &d_alpha,
        d_B, N,                 // d_B: 元のB。転置後は(N x K)となるのでリーディングディメンジョンはN
        d_A, K,                           // d_A: 元のA。転置後は(K x M)となるのでリーディングディメンジョンはK
        &d_beta,
        d_C, N); // d_C: 結果を受け取る行列。row-majorでMxNの行列の場合、転置結果はNxMとなりリーディングディメンジョンはN

    // memcpy to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // comparison C and h_C
    for (int i = 0; i < M * N; ++i)
    {
        if (fabs(C[i] - h_C[i]) > 1e-5)
        {
            std::cout << "Mismatch at index " << i << ": expected " << C[i] << ", got " << h_C[i] << std::endl;
        }
    }

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(A);
    free(B);
    free(C);
    free(h_C);

    return 0;
}